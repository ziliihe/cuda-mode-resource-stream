
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <time.h>

__global__
void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

int main(void)
{
    int N = 1 << 25; // 1M elements

    float *x, *y;

    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
    clock_t start = clock();

    // Run kernel on 1M elements on the CPU
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    add<<<numBlocks, blockSize>>>(N, x, y);
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    clock_t end = clock();
    double cpu_time_used = ((double)(end - start)) / CLOCKS_PER_SEC;
    std::cout << "[ Cuda  ] Time taken: " << cpu_time_used << "s, ";
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}