#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ 
void saxpy(int n, float a, float *x, float *y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        y[i] = a * x[i] + y[i];
}

int main(void)
{
    int N = 1 << 20;
    // 两对数组
    float *x, *y, *d_x, *d_y;
    // x y 指向CPU 内存数组，使用 malloc 分配内存
    x = (float *)malloc(N * sizeof(float));
    y = (float *)malloc(N * sizeof(float));
    // d_x,d_y 指向GPU 内存数组，使用 hipMalloc 分配内存，hipMalloc 是 Cuda 运行时 API
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    // 初始化 x, y CPU 内存数组
    for (int i = 0; i < N; i++)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
    // 将 x, y 拷贝到 GPU 内存数组以初始化
    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);  // 第 4 个参数指明拷贝方向，hipMemcpyHostToDevice： CPU -> GPU; hipMemcpyDeviceToHost： GPU -> CPU
    hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

    // Perform SAXPY on 1M elements
    // 网格中线程块的数量
    // 线程块中线程的数量
    saxpy<<<(N + 255) / 256, 256>>>(N, 2.0f, d_x, d_y);

    hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = max(maxError, abs(y[i] - 4.0f));
    printf("Max error: %f\n", maxError);

    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);
}